#include "hip/hip_runtime.h"
/**
 * Copyright (C) 2022 Samuel Bailey
 */

#include <iostream>
#include "process-life.h"

#define GRID_VALUE(arr, col, row, pitch) \
    (reinterpret_cast<bool *>(reinterpret_cast<char *>(arr) + row*pitch))[col]

__global__
void GridProc::compute_cell(bool *initial_state, size_t initial_pitch, bool *final_state, size_t final_pitch) {
    // For now, assume there are enough processors to have 1 for each cell    
    int column = (blockIdx.x * blockDim.x) + threadIdx.x;
    int row = (blockIdx.y * blockDim.y) + threadIdx.y;

    unsigned int count = 0;
    for (int i = -1; i <= 1; i++) {
        for (int j = -1; j <= 1; j++) {
            // Don't count the middle cell
            if (i == 0 && j == 0) {
                continue;
            }

            count += GRID_VALUE(initial_state, column+i, row+j, initial_pitch);
        }
    }

    // Set the result in the final_state array

    // If there is no item in the cell
    if (GRID_VALUE(initial_state, column, row, initial_pitch) == false) {
        // Spawn a new one
        if (count == 3) {
            GRID_VALUE(final_state, column, row, final_pitch) = true;
        } else { // Else don't
            GRID_VALUE(final_state, column, row, final_pitch) = false;
        }
    } else {
        // There is an item in the cell
        if (count > 1 && count < 4) {
            GRID_VALUE(final_state, column, row, final_pitch) = true;
        } else {
            GRID_VALUE(final_state, column, row, final_pitch) = false;
        }
    }
}

Grid::Grid(int x_size, int y_size, bool *initial_state) {
    h_grid_cols = x_size + 2;
    h_grid_rows = y_size + 2;

    hipMallocPitch(reinterpret_cast<void **>(&d_grid), &d_grid_pitch, h_grid_cols * sizeof(bool), h_grid_rows);
    hipMallocPitch(reinterpret_cast<void **>(&d_next_grid), &d_next_grid_pitch, h_grid_cols * sizeof(bool), h_grid_rows);

    if (initial_state != nullptr) {
        // TODO: Copy data if not nullptr
        std::cout << "copied initial state" << std::endl;
    } else {
        std::cout << "didn't copy initial state" << std::endl;
    }
}

Grid::~Grid() {
    hipFree(d_grid);
    hipFree(d_next_grid);
}

