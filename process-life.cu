#include "hip/hip_runtime.h"
/**
 * Copyright (C) 2022 Samuel Bailey
 */

#include <iostream>
#include <memory>
#include <stdio.h>
#include "process-life.h"
#include "gpu-helpers.h"


__device__
void my_reverse(char str[], int len)
{
    int start, end;
    char temp;
    for(start=0, end=len-1; start < end; start++, end--) {
        temp = *(str+start);
        *(str+start) = *(str+end);
        *(str+end) = temp;
    }
}


__device__
char *my_itoa(int num, char* str, int base)
{
    int i = 0;
    bool isNegative = false;
  
    /* A zero is same "0" string in all base */
    if (num == 0) {
        str[i] = '0';
        str[i + 1] = '\0';
        return str;
    }
  
    /* negative numbers are only handled if base is 10 
       otherwise considered unsigned number */
    if (num < 0 && base == 10) {
        isNegative = true;
        num = -num;
    }
  
    while (num != 0) {
        int rem = num % base;
        str[i++] = (rem > 9)? (rem-10) + 'A' : rem + '0';
        num = num/base;
    }
  
    /* Append negative sign for negative numbers */
    if (isNegative){
        str[i++] = '-';
    }
  
    str[i] = '\0';
 
    my_reverse(str, i);
  
    return str;
}


__device__
int my_min(int a, int b) {
    if (a < b) {
        return a;
    }
    return b;
}


__device__
char *my_strcpy(char *dest, const char *src) {
    int i = 0;
    do {
        dest[i] = src[i];
    } while (src[i++] != 0);
    return dest;
}


#define GRID_VALUE(arr, col, row, pitch) \
    (reinterpret_cast<bool *>(reinterpret_cast<char *>(arr) + row*pitch))[col]

void print() {
    GridProc::gpu_print<<<1, 1>>>();
}

__global__
void GridProc::gpu_print() {
    printf("Hello, world!\n");
}


__global__
void GridProc::compute_cell(bool *initial_state, size_t initial_pitch, bool *final_state, size_t final_pitch, char *instr, char *outstr) {
    // For now, assume there are enough processors to have 1 for each cell
    // Need to start at [1, 1] instead of [0, 0] because of the boundaries
    int column = (blockIdx.x * blockDim.x) + threadIdx.x + 1;
    int row = (blockIdx.y * blockDim.y) + threadIdx.y + 1;

    if (threadIdx.x == 0 && threadIdx.y == 0 && blockIdx.x == 0 && blockIdx.y == 0) {
        char tmp[] = "It's working!!! NAAAT.";
        my_strcpy(outstr, tmp);
    }
    return;

    unsigned int count = 0;
    for (int j = -1; j <= 1; j++) {
        for (int i = -1; i <= 1; i++) {
            // Don't count the middle cell
            if (i == 0 && j == 0) {
                continue;
            }
            if (threadIdx.x == 0 && threadIdx.y == 0) {
                // printf("Block: %d, Thread: %d, reading from (%d, %d)\n", (int)threadIdx.x, (int)threadIdx.y, column+i, row+j);
            }

            count += GRID_VALUE(initial_state, column+i, row+j, initial_pitch);
        }
    }
    return;

    // Set the result in the final_state array

    // If there is no item in the cell
    if (GRID_VALUE(initial_state, column, row, initial_pitch) == false) {
        // Spawn a new one
        if (count == 3) {
            GRID_VALUE(final_state, column, row, final_pitch) = true;
        } else { // Else don't
            GRID_VALUE(final_state, column, row, final_pitch) = false;
        }
    } else {
        // There is an item in the cell
        if (count > 1 && count < 4) {
            GRID_VALUE(final_state, column, row, final_pitch) = true;
        } else {
            GRID_VALUE(final_state, column, row, final_pitch) = false;
        }
    }
}

Grid::Grid(int x_size, int y_size, bool *initial_state) {
    h_grid_cols = x_size + 2;
    h_grid_rows = y_size + 2;

    hipMalloc(reinterpret_cast<void **>(&d_instr), 100 * sizeof(char));
    hipHostMalloc(reinterpret_cast<void **>(&h_instr), 100 * sizeof(char));
    hipMalloc(reinterpret_cast<void **>(&d_outstr), 100 * sizeof(char));
    hipHostMalloc(reinterpret_cast<void **>(&h_outstr), 100 * sizeof(char));

    // Malloc grids
    hipMallocPitch(reinterpret_cast<void **>(&d_grid), &d_grid_pitch, h_grid_cols * sizeof(bool), h_grid_rows);
    hipMallocPitch(reinterpret_cast<void **>(&d_next_grid), &d_next_grid_pitch, h_grid_cols * sizeof(bool), h_grid_rows);
    hipHostMalloc(&h_grid, h_grid_rows * h_grid_cols * sizeof(bool));

    // Clear grids
    hipMemset2D(d_grid, d_grid_pitch, false, h_grid_cols * sizeof(bool), h_grid_rows);
    hipMemset2D(d_next_grid, d_grid_pitch, false, h_grid_cols * sizeof(bool), h_grid_rows);
    hipMemset(h_grid, false, h_grid_rows * h_grid_cols * sizeof(bool));

    // Populate grids
    if (initial_state != nullptr) {
        
        // First copy into h_grid, which has the same dimensions as d_grid
        for (int j = 0; j < y_size; j++) {
            // Copy each row using memcpy
            memcpy(&h_grid[1 + (j+1)*h_grid_cols], &initial_state[j * x_size], x_size);
        }

        // Copy host grid to the device
        hipMemcpy2D(d_grid, d_grid_pitch, initial_state, h_grid_cols * sizeof(bool), h_grid_cols, h_grid_rows, hipMemcpyHostToDevice);

        std::cout << "copied initial state" << std::endl;
    } else {
        std::cout << "didn't copy initial state" << std::endl;
    }
}

Grid::~Grid() {
    hipFree(d_outstr);
    hipHostFree(h_outstr);

    hipFree(d_grid);
    hipFree(d_next_grid);
    hipHostFree(h_grid);
    std::cout << "Freed grid" << std::endl;
    std::cout << "Sizeof bool = " << sizeof(bool) << std::endl;
}

void Grid::step_forwards(int n_steps) {
    auto thread_dims = dim3(h_grid_cols, h_grid_rows);

    for (int i = 0; i < n_steps; i++) {
        // Perform a step
        GridProc::compute_cell<<<1, thread_dims>>>(d_grid, d_grid_pitch, d_next_grid, d_next_grid_pitch, d_instr, d_outstr);
        hipMemcpy(h_outstr, d_outstr, 100, hipMemcpyDeviceToHost);

        std::cout << "OUTSTR: " << h_outstr << std::endl;
        // TODO: copy boundaries

        // swap pointers
        bool *tmp_ptr = d_grid;
        d_grid = d_next_grid;
        d_next_grid = tmp_ptr;
    }
}

void Grid::update_host_grid() {
    hipMemcpy2D(h_grid, h_grid_cols * sizeof(bool), d_grid, d_grid_pitch, h_grid_cols * sizeof(bool), h_grid_rows, hipMemcpyDeviceToHost);
}

std::unique_ptr<bool[]> Grid::get_host_grid(bool reallign) {
    if (!reallign) {
        auto result_grid = std::make_unique<bool[]>(h_grid_cols*h_grid_rows);
        memcpy(result_grid.get(), h_grid, h_grid_cols*h_grid_rows*sizeof(bool));
        return std::move(result_grid);
    }

    // Reallign
    auto result_grid = std::make_unique<bool[]>((h_grid_cols-2) * (h_grid_rows-2));
    for (int j = 1; j < h_grid_rows-1; j++) {
        memcpy(&result_grid[(j-1) * (h_grid_cols-2)], &h_grid[1 + (j * h_grid_cols)], (h_grid_cols-2)*sizeof(bool));
    }
    return std::move(result_grid);
}