#include "process-life.h"

Grid::Grid(int x_size, int y_size) {

}

Grid::Grid(int x_size, int y_size, bool **initial_state) {
    void *d_grid_tmp;
    hipMallocPitch(&d_grid_tmp, &d_grid_pitch, x_size, y_size);
    d_grid = static_cast<bool *>(d_grid_tmp);
}

Grid::~Grid() {
    hipFree(d_grid);
}