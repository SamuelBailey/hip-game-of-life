#include "hip/hip_runtime.h"
#include <iostream>
#include "process-life.h"

__global__
void GridProc::compute_cell(bool *initial_state, bool *final_state);

Grid::Grid(int x_size, int y_size, bool *initial_state) {
    void *d_grid_tmp;
    hipMallocPitch(&d_grid_tmp, &d_grid_pitch, x_size * sizeof(bool), y_size);
    d_grid = static_cast<bool *>(d_grid_tmp);

    hipMallocPitch(&d_grid_tmp, &d_next_grid_pitch, x_size * sizeof(bool), y_size);
    d_next_grid = static_cast<bool *>(d_grid_tmp);

    if (initial_state != nullptr) {
        // TODO: Copy data if not nullptr
        std::cout << "copied initial state" << std::endl;
    } else {
        std::cout << "didn't copy initial state" << std::endl;
    }
}

Grid::~Grid() {
    hipFree(d_grid);
    hipFree(d_next_grid);
}

