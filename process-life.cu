#include "process-life.h"

Grid::Grid(int x_size, int y_size) {

}

Grid::Grid(int x_size, int y_size, bool **initial_state) {
    hipChannelFormatDesc desc = {
        .x = x_size,
        .y = y_size,
        .z = 0,
        .w = 0,
        .f = hipChannelFormatKind::hipChannelFormatKindUnsigned
    };
    // hipMalloc3D()
}